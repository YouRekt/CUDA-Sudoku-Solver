#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <fstream>

#ifdef __INTELLISENSE__
#include "intelisense.h" // Fixes intellisense not recognizing atomicAdd();
#endif

#define CUDACheckError(cudaStatus) \
	if(cudaStatus != hipSuccess) {	\
		std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - "; \
		std::cerr << hipGetErrorName(cudaStatus) << ": " << hipGetErrorString(cudaStatus) << std::endl; \
	} \

#define THREADS_PER_BLOCK 1024 // Anything above just breaks
#define NUM_OF_BLOCKS 128

#define MAX_BOARDS (INT_MAX >> 4)
#define BFS_DEPTH 30
#define N 9
#define SQRT_N 3
#define BOARD_SIZE 81

// Host helper functions
void printDeviceArray(const char* name, int* d_array, int size) {
	int* h_array = new int[size];
	CUDACheckError(hipMemcpy(h_array, d_array, size * sizeof(int), hipMemcpyDeviceToHost));

	std::cout << "\n" << name << ":\n";
	for (int i = 0; i < size; i++) {
		if (i % BOARD_SIZE == 0 && size > 1) {
			std::cout << "#" << (i / BOARD_SIZE) + 1 << std::endl;
		}

		std::cout << h_array[i] << " ";

		// Add a newline after every row
		if ((i + 1) % N == 0) {
			std::cout << "\n";
		}

		// Add an extra newline after printing a full board (NxN grid)
		if ((i + 1) % BOARD_SIZE == 0) {
			std::cout << std::endl;
		}
	}
	delete[] h_array;
}

int countEmptySpaces(int* board) {
	int count = 0;
	for (int i = 0; i < BOARD_SIZE; i++) {
		if (board[i] == 0)
			count++;
	}
	return count;
}

void print(int* board) {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++)
			std::cout << board[i * N + j] << " ";
		std::cout << std::endl;
	}
}

// Device helper functions
__device__ void printBoard(int* board, int thread) {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			printf("Thread %d: [%d,%d] = %d\n", thread, i, j, board[N * i + j]);
		}
		printf("\n");
	}
	printf("\n");
}

__device__ void clearSeen(bool* seen) {
	memset(seen, false, sizeof(bool) * N);
}

__device__ void copyBoard(const int* src, int* dst) {
	for (int i = 0; i < BOARD_SIZE; i++) {
		dst[i] = src[i];
	}
}

__device__ void initializeEmptyCellIndexes(int* board, int* emptyCellIndexes, int* emptyCellCounts) {
	int index = 0;
	for (int i = 0; i < BOARD_SIZE; i++) {
		if (board[i] == 0) {
			emptyCellIndexes[index] = i;
			index++;
		}
	}
	*emptyCellCounts = index;
}

__device__ bool isValid(int* d_board, int changedIndex) { // DFS version
	int changedRow = changedIndex / N;
	int changedColumn = changedIndex % N;

	if (d_board[changedIndex] < 1 || d_board[changedIndex]>9)
		return false;

	bool seen[N];

	// Check column
	clearSeen(seen);
	for (int i = 0; i < N; i++) {
		int number = d_board[i * N + changedColumn] - 1;
		if (number >= 0) {
			if (seen[number])
				return false;
			seen[number] = true;
		}
	}

	// Check row
	clearSeen(seen);
	for (int i = 0; i < N; i++) {
		int number = d_board[changedRow * N + i] - 1;
		if (number >= 0) {
			if (seen[number])
				return false;
			seen[number] = true;
		}
	}

	// Check box
	clearSeen(seen);
	int b = (SQRT_N * (changedRow / SQRT_N)) + (changedColumn / SQRT_N);
	for (int i = 0; i < SQRT_N; i++) {
		for (int j = 0; j < SQRT_N; j++) {
			int b_row = b / SQRT_N, b_col = b % SQRT_N;
			int number = d_board[(SQRT_N * b_row + i) * N + SQRT_N * b_col + j] - 1;
			if (number >= 0) {
				if (seen[number])
					return false;
				seen[number] = true;
			}
		}
	}

	return true;
}

__device__ bool isValid(int* d_board, int changedIndex, int value) { // BFS version
	int changedRow = changedIndex / N;
	int changedColumn = changedIndex % N;

	bool seen[N];

	// Check column
	clearSeen(seen);
	for (int i = 0; i < N; i++) {
		int number = d_board[i * N + changedColumn] - 1;
		if (number >= 0) {
			if (seen[number])
				return false;
			seen[number] = true;
		}
	}
	if (seen[value - 1])
		return false;

	// Check row
	clearSeen(seen);
	for (int i = 0; i < N; i++) {
		int number = d_board[changedRow * N + i] - 1;
		if (number >= 0) {
			if (seen[number])
				return false;
			seen[number] = true;
		}
	}
	if (seen[value - 1])
		return false;

	// Check box
	clearSeen(seen);
	int b = SQRT_N * (changedRow / SQRT_N) + changedColumn / SQRT_N;
	for (int i = 0; i < SQRT_N; i++) {
		for (int j = 0; j < SQRT_N; j++) {
			int b_row = b / SQRT_N, b_col = b % SQRT_N;
			int number = d_board[(SQRT_N * b_row + i) * N + SQRT_N * b_col + j] - 1;
			if (number >= 0) {
				if (seen[number])
					return false;
				seen[number] = true;
			}
		}
	}
	if (seen[value - 1])
		return false;

	return true;
}

__global__ void DFS(int* BFSearchedBoards, const int boardsCount, int* emptyCellIndexes, int* emptyCellCounts, bool* finished, int* solvedBoard) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	while (!*finished && index < boardsCount) {
		int* currentBoard = BFSearchedBoards + index * BOARD_SIZE;
		int* currentEmptyCells = emptyCellIndexes + index * BOARD_SIZE;
		int emptyCellIndex = 0;

		while (emptyCellIndex >= 0 && emptyCellIndex < emptyCellCounts[index]) {
			currentBoard[currentEmptyCells[emptyCellIndex]]++;

			if (currentBoard[currentEmptyCells[emptyCellIndex]] > 9) {
				currentBoard[currentEmptyCells[emptyCellIndex]] = 0;
				emptyCellIndex--;  // Backtrack
			}
			else if (isValid(currentBoard, currentEmptyCells[emptyCellIndex])) {
				emptyCellIndex++;  // Move forward
			}
		}

		if (emptyCellIndex == emptyCellCounts[index]) {
			*finished = true;
			copyBoard(currentBoard, solvedBoard);
		}

		index += gridDim.x * blockDim.x; // This ensures we dont mess up the work of other threads
	}
}

__global__ void BFS(int* BFSearchedBoards, int startIndex, int cutoffIndex, int* nextBoardIndex, int* emptyCellIndexes, int* emptyCellCounts) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	while (index < cutoffIndex - startIndex) {
		bool found = false;
		int offsetIndex = index + startIndex;

		for (int i = BOARD_SIZE * offsetIndex; i < BOARD_SIZE * (offsetIndex + 1) && !found; i++) {
			if (BFSearchedBoards[i] == 0) {
				found = true;
				int normalizedIndex = i - BOARD_SIZE * offsetIndex;

				for (int val = 1; val <= N; val++) { // Test each value for the empty cell and if it's valid add it to the boards array
					if (isValid(BFSearchedBoards + BOARD_SIZE * offsetIndex, normalizedIndex, val)) {
						int boardIndex = atomicAdd(nextBoardIndex, 1); // This ensures contingency of the boards array
						copyBoard(BFSearchedBoards + BOARD_SIZE * offsetIndex, BFSearchedBoards + BOARD_SIZE * boardIndex);
						BFSearchedBoards[BOARD_SIZE * boardIndex + normalizedIndex] = val;

						initializeEmptyCellIndexes(BFSearchedBoards + BOARD_SIZE * boardIndex, emptyCellIndexes + BOARD_SIZE * boardIndex, emptyCellCounts + boardIndex); // Update the information about empty cells for DFS
					}
				}
			}
		}

		index += blockDim.x * gridDim.x; // This ensures we dont mess up the work of other threads
	}
}

int main() {
	int* h_board = new int[BOARD_SIZE];

	std::ifstream in("easy.txt");
	char c;
	int i = 0;
	while (in >> c && i < BOARD_SIZE) {
		h_board[i] = c - '0';
		i++;
	}
	in.close();

	int* d_BFSearchedBoards;
	int* d_emptyCellCounts;
	int* d_emptyCellIndexes;
	int* d_nextBoardIndex;

	CUDACheckError(hipMalloc((void**)&d_BFSearchedBoards, MAX_BOARDS * sizeof(int)));
	CUDACheckError(hipMalloc((void**)&d_emptyCellIndexes, MAX_BOARDS * sizeof(int)));
	CUDACheckError(hipMalloc((void**)&d_emptyCellCounts, (MAX_BOARDS / BOARD_SIZE + 1) * sizeof(int)));
	CUDACheckError(hipMalloc((void**)&d_nextBoardIndex, sizeof(int)));

	CUDACheckError(hipMemset(d_BFSearchedBoards, 0, MAX_BOARDS * sizeof(int)));
	CUDACheckError(hipMemset(d_emptyCellIndexes, 0, MAX_BOARDS * sizeof(int)));

	// I tried calling hipMemset(d_nextBoardIndex,1,sizeof(int)) but it outputs 16843009. Reason: https://forums.developer.nvidia.com/t/can-we-use-memset-for-non-zero-initial-value/4032
	int initValue = 1; // Hence this is a simple workaround.
	CUDACheckError(hipMemcpy(d_nextBoardIndex, &initValue, sizeof(int), hipMemcpyHostToDevice));

	// Copy the initial board to BFS boards
	CUDACheckError(hipMemcpy(d_BFSearchedBoards, h_board, BOARD_SIZE * sizeof(int), hipMemcpyHostToDevice));

	// Timing variables
	hipEvent_t start, stop;
	float milliseconds = 0;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	int startIndex = 0;
	int cutoffIndex = 1;

	BFS << <NUM_OF_BLOCKS, THREADS_PER_BLOCK >> > (d_BFSearchedBoards, startIndex, cutoffIndex, d_nextBoardIndex, d_emptyCellIndexes, d_emptyCellCounts);

	int depth;

	if (countEmptySpaces(h_board) > BOARD_SIZE - 2 * N) { // If the board is almost empty it is actually faster to skip the BFS
		depth = 0;
	}
	else {
		depth = BFS_DEPTH;
	}

	// Generate more boards with more iterations of BFS
	for (int i = 0; i < depth; i++) {
		startIndex = cutoffIndex;
		CUDACheckError(hipMemcpy(&cutoffIndex, d_nextBoardIndex, sizeof(int), hipMemcpyDeviceToHost));
		BFS << <NUM_OF_BLOCKS, THREADS_PER_BLOCK >> > (d_BFSearchedBoards, startIndex, cutoffIndex, d_nextBoardIndex, d_emptyCellIndexes, d_emptyCellCounts);
	}
	startIndex = cutoffIndex;
	CUDACheckError(hipMemcpy(&cutoffIndex, d_nextBoardIndex, sizeof(int), hipMemcpyDeviceToHost));

	int boardsCount = cutoffIndex - startIndex;
	bool* d_finished;
	int* d_solvedBoard;
	int* h_solvedBoard = new int[BOARD_SIZE];
	bool* h_finished = new bool;

	CUDACheckError(hipMalloc((void**)&d_finished, sizeof(bool)));
	CUDACheckError(hipMalloc((void**)&d_solvedBoard, BOARD_SIZE * sizeof(int)));

	CUDACheckError(hipMemset(d_finished, false, sizeof(bool)));

	DFS << <NUM_OF_BLOCKS, THREADS_PER_BLOCK >> > (d_BFSearchedBoards + BOARD_SIZE * startIndex, boardsCount, d_emptyCellIndexes + BOARD_SIZE * startIndex, d_emptyCellCounts + startIndex, d_finished, d_solvedBoard);

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&milliseconds, start, stop);

	std::cout << "Execution Time: " << milliseconds << " ms" << std::endl;

	CUDACheckError(hipMemcpy(h_finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));
	if (*h_finished) {
		CUDACheckError(hipMemcpy(h_solvedBoard, d_solvedBoard, BOARD_SIZE * sizeof(int), hipMemcpyDeviceToHost));
		print(h_solvedBoard);
	}
	else
		printf("No solutuion found");

	hipEventDestroy(start);
	hipEventDestroy(stop);
	delete[] h_board;
	delete[] h_solvedBoard;
	delete h_finished;
	hipFree(d_BFSearchedBoards);
	hipFree(d_emptyCellCounts);
	hipFree(d_emptyCellIndexes);
	hipFree(d_nextBoardIndex);
	hipFree(d_finished);
	hipFree(d_solvedBoard);

	return 0;
}
